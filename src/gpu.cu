#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples
#include <hip/device_functions.h> // helper functions for SDK examples

#include "gpu.h"

using namespace std;



void printCard(){
    const int kb = 1024;
    const int mb = kb * kb;
    cout << "NBody.GPU" << endl << "=========" << endl << endl;

    cout << "CUDA version:   v" << CUDART_VERSION << endl;    

    int devCount;
    hipGetDeviceCount(&devCount);
    wcout << "CUDA Devices: " << endl << endl;

    for(int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        cout << i << ": " << props.name << ": " << props.major << "." << props.minor << endl;
        cout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
        cout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
        cout << "  Constant memory: " << props.totalConstMem / kb << "kb" << endl;
        cout << "  Block registers: " << props.regsPerBlock << endl << endl;

        cout << "  Warp size:         " << props.warpSize << endl;
        cout << "  Threads per block: " << props.maxThreadsPerBlock << endl;
        cout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1]  << ", " << props.maxThreadsDim[2] << " ]" << endl;
        cout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1]  << ", " << props.maxGridSize[2] << " ]" << endl;
        cout << endl;
    }
}

void allocateDetector(deviceDetector* d){
     if(hipSuccess != hipMalloc((void**)&d->sstrips, sizeof(bool)*SIZE_LAYER*NB_LAYER))
        cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
 
     // max of MAX_NB_STUBS_PER_SSTRIPS stubs per superstrip
     if(hipSuccess != hipMalloc((void**)&d->stubs, sizeof(int)*SIZE_LAYER*NB_LAYER*MAX_NB_STUBS_PER_SSTRIPS))
        cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;

     resetDetector(d);
}

void resetDetector(deviceDetector* d, hipStream_t* stream){
  if(stream==NULL){
     if(hipSuccess != hipMemset(d->sstrips,0,sizeof(bool)*SIZE_LAYER*NB_LAYER))
        cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;	
     if(hipSuccess != hipMemset(d->stubs, -1, sizeof(int)*SIZE_LAYER*NB_LAYER*MAX_NB_STUBS_PER_SSTRIPS))
        cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;	  
  }
  else{
     if(hipSuccess != hipMemsetAsync(d->sstrips,0,sizeof(bool)*SIZE_LAYER*NB_LAYER, *stream))
        cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;	
     if(hipSuccess != hipMemsetAsync(d->stubs, -1, sizeof(int)*SIZE_LAYER*NB_LAYER*MAX_NB_STUBS_PER_SSTRIPS, *stream))
        cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;	  
  }
}

void freeDetector(deviceDetector* d){
     deleteTimer();
     hipFree(d->sstrips);
     hipFree(d->stubs);
}

void allocateBank(patternBank* p, int nbPatterns){
     if(hipSuccess != hipMalloc((void**)&p->banks, sizeof(unsigned int)*PATTERN_SIZE*nbPatterns))
       cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
     if(hipSuccess != hipMalloc((void**)&p->nb_patterns, sizeof(int)))
       cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
     if(hipSuccess !=hipMemset(p->banks,255, sizeof(unsigned int)*PATTERN_SIZE*nbPatterns))
       cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
     if(hipSuccess !=hipMemset(p->nb_patterns,nbPatterns, sizeof(int)))
       cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
}

void freeBank(patternBank* p){
     hipFree(p->banks);
     hipFree(p->nb_patterns);
}

void allocateStubs(deviceStubs* s){
 if(hipSuccess != hipMalloc((void**)&s->stubs, sizeof(char)*CUDA_MAX_NB_STUBS*CUDA_STUB_SIZE))
       cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
 if(hipSuccess != hipMalloc((void**)&s->active_stubs, sizeof(bool)*CUDA_MAX_NB_STUBS))
       cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
 if(hipSuccess != hipMalloc((void**)&s->nb_stubs, sizeof(int)))
       cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
}

void freeStubs(deviceStubs* s){
     hipFree(s->stubs);
     hipFree(s->active_stubs);
     hipFree(s->nb_stubs);
}

void allocateParameters(deviceParameters* dp){
  if(hipSuccess != hipMalloc((void**)&dp->result, sizeof(int)))
    cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;

  if(hipSuccess != hipMalloc((void**)&dp->threshold, sizeof(int)))
    cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
 
  if(hipSuccess != hipMalloc((void**)&dp->iter, sizeof(int)))
    cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
 
  if(hipSuccess != hipMalloc((void**)&dp->nbPatterns, sizeof(int)))
    cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
}

void freeParameters(deviceParameters* dp){
  hipFree(dp->result);
  hipFree(dp->threshold);
  hipFree(dp->iter);
  hipFree(dp->nbPatterns);
}

void cudaSetLink(patternBank* p, int index, unsigned int* vals){
  if(hipSuccess != hipMemcpy(p->banks+index,vals,PATTERN_LAYERS*PATTERN_SSTRIPS*sizeof(unsigned int), hipMemcpyHostToDevice))
      cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
}

void cudaSetNbPatterns(patternBank* p, int nb){
  if(hipSuccess != hipMemcpy(p->nb_patterns,&nb,sizeof(int), hipMemcpyHostToDevice))
      cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl; 
}

void cudaCopyStubs(char* stubs, deviceStubs* d_stubs, int nb, hipStream_t* stream){
 if(stream==NULL){
     if(hipSuccess != hipMemcpy(d_stubs->stubs,stubs,nb*CUDA_STUB_SIZE*sizeof(char), hipMemcpyHostToDevice))
          cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
     if(hipSuccess !=hipMemset(d_stubs->active_stubs,false, nb*sizeof(bool)))
          cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
 }
 else{
     if(hipSuccess != hipMemcpyAsync(d_stubs->stubs,stubs,nb*CUDA_STUB_SIZE*sizeof(char), hipMemcpyHostToDevice, *stream))
          cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
     if(hipSuccess !=hipMemsetAsync(d_stubs->active_stubs,false, nb*sizeof(bool), *stream))
          cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
 }
}

void cudaGetActiveStubs(bool* active_stubs, deviceStubs* d_stubs, int* nb, hipStream_t* stream){
  if(stream==NULL){
    if(hipSuccess != hipMemcpy(active_stubs,d_stubs->active_stubs,(*nb)*sizeof(bool), hipMemcpyDeviceToHost))
        cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
  }
  else{
    if(hipSuccess != hipMemcpyAsync(active_stubs,d_stubs->active_stubs,(*nb)*sizeof(bool), hipMemcpyDeviceToHost, *stream))
        cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
  }
}

void cudaShowBank(patternBank*p){
     unsigned int val;
     for(int i=0;i<10;i++){
       for(int j=0;j<8;j++){
         for(int k=0;k<8;k++){
           if(hipSuccess != hipMemcpy(&val,p->banks+i*PATTERN_SIZE+j*PATTERN_LAYERS+k,sizeof(unsigned int), hipMemcpyDeviceToHost))
             cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
           cout<<val<<" ";
	 }
	 cout<<endl;
       }
       cout<<endl;
     }
     cout<<endl;
     cout<<endl;
}

void cudaShowStubs(deviceStubs* s, int nb){
     char val;
     for(int i=0;i<nb;i++){
       if(hipSuccess != hipMemcpy(&val,s->stubs+i*CUDA_STUB_SIZE+0,sizeof(char), hipMemcpyDeviceToHost))
         cout<<"cudaShowStubs : error! "<<hipGetErrorString(hipGetLastError())<<endl;
       cout<<"layer "<<(int)val<<" ";
       if(hipSuccess != hipMemcpy(&val,s->stubs+i*CUDA_STUB_SIZE+1,sizeof(char), hipMemcpyDeviceToHost))
         cout<<"cudaShowStubs : error! "<<hipGetErrorString(hipGetLastError())<<endl;
       cout<<"ladder "<<(int)val<<" ";
       if(hipSuccess != hipMemcpy(&val,s->stubs+i*CUDA_STUB_SIZE+2,sizeof(char), hipMemcpyDeviceToHost))
         cout<<"cudaShowStubs : error! "<<hipGetErrorString(hipGetLastError())<<endl;
       cout<<"module "<<(int)val<<" ";
       if(hipSuccess != hipMemcpy(&val,s->stubs+i*CUDA_STUB_SIZE+3,sizeof(char), hipMemcpyDeviceToHost))
         cout<<"cudaShowStubs : error! "<<hipGetErrorString(hipGetLastError())<<endl;
       cout<<"segment "<<(int)val<<" ";
       if(hipSuccess != hipMemcpy(&val,s->stubs+i*CUDA_STUB_SIZE+4,sizeof(char), hipMemcpyDeviceToHost))
         cout<<"cudaShowStubs : error! "<<hipGetErrorString(hipGetLastError())<<endl;
       cout<<"superstrip "<<(int)val<<" ";

       cout<<endl;
     }
}

void resetCard(){
    if(hipSuccess != hipDeviceReset())
      cout<<"Error : "<<hipGetErrorString(hipGetLastError())<<endl;
}

StopWatchInterface *theTimer=0;

void initialiseTimer()
{
  theTimer=0;
  sdkCreateTimer(&theTimer);
}

void startTimer()
{
   sdkResetTimer(&theTimer);
   sdkStartTimer(&theTimer);
}

float stopTimer()
{
  sdkStopTimer(&theTimer);
  float t=sdkGetTimerValue(&theTimer);
  printf("Processing time: %f (ms)\n",t );
  return t;
}

void deleteTimer()
{
   sdkDeleteTimer(&theTimer);
}

__global__ void cudaSetHits(char* d_stubs, int nbStubs, bool* d_det, int* d_det_stubs){
  int index = threadIdx.x * CUDA_STUB_SIZE + blockIdx.x * blockDim.x * CUDA_STUB_SIZE;

  if(index/CUDA_STUB_SIZE<nbStubs){
    int detIndex = d_stubs[index]*SIZE_LAYER+d_stubs[index+1]*SIZE_LADDER+d_stubs[index+2]*SIZE_MODULE+d_stubs[index+3]*SIZE_SEGMENT+d_stubs[index+4];
    d_det[detIndex]=1;
    int det_stubs_index=detIndex*MAX_NB_STUBS_PER_SSTRIPS;

    for(int count=0;count<MAX_NB_STUBS_PER_SSTRIPS;count++){
      int old = atomicCAS(&d_det_stubs[det_stubs_index+count],-1,index);
      if(old==-1)
        break;
    }
  }
}

__global__ void cudaGetActivePatterns(bool* detector, int* detector_stubs, bool* active_stubs, unsigned int* patterns, int* threshold, int* nbIter, int* nbMaxPatterns, int* nbActivePatterns){
   int index = blockIdx.x * blockDim.x * PATTERN_SIZE * (*nbIter) + threadIdx.x * PATTERN_SIZE * (*nbIter);
   char score = 0;

   for(int l=0;l<(*nbIter);l++){
     if(index<PATTERN_SIZE*(*nbMaxPatterns)){
       score = 0;
       for(int i=0;i<PATTERN_LAYERS;i++){
         for(int j=0;j<PATTERN_SSTRIPS;j++){
           unsigned int ref = patterns[index+i*PATTERN_SSTRIPS+j];
           if(ref==PATTERN_UNUSED)
             break;
           if(detector[ref]){
             score++;
             break;
           }
         }
       }
       if(score>=(*threshold)){
         atomicAdd(nbActivePatterns,1);
         for(int i=0;i<PATTERN_LAYERS;i++){
           for(int j=0;j<PATTERN_SSTRIPS;j++){
             unsigned int ref = patterns[index+i*PATTERN_SSTRIPS+j];
             if(ref==PATTERN_UNUSED)
               break;
             int stub_index = ref*MAX_NB_STUBS_PER_SSTRIPS; 
	     for(int k=0;k<MAX_NB_STUBS_PER_SSTRIPS;k++){
	       if(detector_stubs[stub_index+k]!=-1){
		 active_stubs[detector_stubs[stub_index+k]/CUDA_STUB_SIZE]=1;
               }
 	     }
           }
         }
       }
     }
     index += PATTERN_SIZE;
   }
   // __syncthreads();
}

void cudaSetHitsWrapper(deviceStubs* d_stubs, int nbStubs, deviceDetector* d_det, hipStream_t* stream){
  if(nbStubs>0){
    cudaSetHits<<<nbStubs,1,0,((stream==NULL)?0:*stream)>>>(d_stubs->stubs,nbStubs,d_det->sstrips,d_det->stubs);
  }
}

void getHitsArray(deviceDetector* det, int* list, int nb){
   if(hipSuccess !=hipMemcpy(list,det->stubs,  sizeof(int)*SIZE_LAYER*NB_LAYER*MAX_NB_STUBS_PER_SSTRIPS,hipMemcpyDeviceToHost))
       cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
}

int cudaGetActivePatternsWrapper(deviceDetector* detector, patternBank* patterns, deviceStubs* stubs, deviceParameters* params, int threshold, int nbThreads, int nbBlocks, hipStream_t* stream){

  if(stream==NULL){
    if(hipSuccess !=hipMemset(params->result,0, sizeof(int)))
         cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
  }
  else{
    if(hipSuccess !=hipMemsetAsync(params->result,0, sizeof(int),*stream))
         cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
  }

    if(stream==NULL)
        cudaGetActivePatterns<<<nbBlocks,nbThreads>>>(detector->sstrips, detector->stubs, stubs->active_stubs, patterns->banks, params->threshold, params->iter, params->nbPatterns, params->result);
    else
        cudaGetActivePatterns<<<nbBlocks,nbThreads,0,*stream>>>(detector->sstrips, detector->stubs, stubs->active_stubs, patterns->banks, params->threshold, params->iter, params->nbPatterns, params->result);
  int res=0;

  if(stream==NULL){
    if(hipSuccess != hipMemcpy(&res,params->result,sizeof(int), hipMemcpyDeviceToHost))
      cout<<"error! "<<hipGetErrorString(hipGetLastError())<<endl;
  }

  return res;
}
